#include "cuda_functions.cuh"

hipError_t element_step_with_CUDA(Eigen::VectorXd * u, Eigen::VectorXd * v, Eigen::VectorXd * a,
	Eigen::VectorXd * load, Eigen::VectorXd * supports, int * neighbors, Eigen::MatrixXd * K,
	Eigen::MatrixXd * C, Eigen::MatrixXd * M, Eigen::Matrix2d Kc, int n_elems, int n_nodes)
{
	// Eigen structures will be copied into arrays of doubles
	double * u_;
	Eigen::Map<Eigen::VectorXd>(u_, u->rows(), u->cols()); // Layout is column by column
}
